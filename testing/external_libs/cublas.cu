#include <unittest/unittest.h>
#include <cusp/array2d.h>
#include <cusp/gallery/poisson.h>
#include <cusp/blas/cublas/blas.h>
#include <cusp/blas/blas.h>

void TestCUBLASamax(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(6);
    View view_x(x);

    x[0] =  7.0f;
    x[1] = -5.0f;
    x[2] =  4.0f;
    x[3] = -3.0f;
    x[4] =  0.0f;
    x[5] =  1.0f;

    ASSERT_EQUAL(cusp::blas::amax(cublas,x), 0);

    ASSERT_EQUAL(cusp::blas::amax(cublas,view_x), 0);

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLASamax);

void TestCUBLASasum(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(6);
    View view_x(x);

    x[0] =  7.0f;
    x[1] =  5.0f;
    x[2] =  4.0f;
    x[3] = -3.0f;
    x[4] =  0.0f;
    x[5] =  1.0f;

    ASSERT_EQUAL(cusp::blas::asum(cublas,x), 20.0f);

    ASSERT_EQUAL(cusp::blas::asum(cublas,view_x), 20.0f);

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLASasum);

void TestCUBLASaxpy(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(4);
    Array y(4);

    x[0] =  7.0f;
    y[0] =  0.0f;
    x[1] =  5.0f;
    y[1] = -2.0f;
    x[2] =  4.0f;
    y[2] =  0.0f;
    x[3] = -3.0f;
    y[3] =  5.0f;

    cusp::blas::axpy(cublas, x, y, 2.0f);

    ASSERT_EQUAL(y[0],  14.0);
    ASSERT_EQUAL(y[1],   8.0);
    ASSERT_EQUAL(y[2],   8.0);
    ASSERT_EQUAL(y[3],  -1.0);

    View view_x(x);
    View view_y(y);

    cusp::blas::axpy(cublas, view_x, view_y, 2.0f);

    ASSERT_EQUAL(y[0],  28.0);
    ASSERT_EQUAL(y[1],  18.0);
    ASSERT_EQUAL(y[2],  16.0);
    ASSERT_EQUAL(y[3],  -7.0);

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLASaxpy);

void TestCUBLAScopy(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(4);

    x[0] =  7.0f;
    x[1] =  5.0f;
    x[2] =  4.0f;
    x[3] = -3.0f;

    {
        Array y(4, -1);
        cusp::blas::copy(cublas, x, y);
        ASSERT_EQUAL(x, y);
    }

    {
        Array y(4, -1);
        View view_x(x);
        View view_y(y);
        cusp::blas::copy(cublas, view_x, view_y);
        ASSERT_EQUAL(x, y);
    }

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLAScopy);

void TestCUBLASdot(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(6);
    Array y(6);

    x[0] =  7.0f;
    y[0] =  0.0f;
    x[1] =  5.0f;
    y[1] = -2.0f;
    x[2] =  4.0f;
    y[2] =  0.0f;
    x[3] = -3.0f;
    y[3] =  5.0f;
    x[4] =  0.0f;
    y[4] =  6.0f;
    x[5] =  4.0f;
    y[5] =  1.0f;

    ASSERT_EQUAL(cusp::blas::dot(cublas, x, y), -21.0f);

    View view_x(x);
    View view_y(y);
    ASSERT_EQUAL(cusp::blas::dot(cublas, view_x, view_y), -21.0f);

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLASdot);

void TestCUBLASnrm2(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(6);

    x[0] =  7.0f;
    x[1] =  5.0f;
    x[2] =  4.0f;
    x[3] = -3.0f;
    x[4] =  0.0f;
    x[5] =  1.0f;

    ASSERT_EQUAL(cusp::blas::nrm2(cublas, x), 10.0f);

    ASSERT_EQUAL(cusp::blas::nrm2(cublas, View(x)), 10.0f);
}
DECLARE_UNITTEST(TestCUBLASnrm2);

void TestCUBLASscal(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array1d<float, MemorySpace>       Array;
    typedef typename cusp::array1d<float, MemorySpace>::view View;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array x(6);

    x[0] =  7.0f;
    x[1] =  5.0f;
    x[2] =  4.0f;
    x[3] = -3.0f;
    x[4] =  0.0f;
    x[5] =  4.0f;

    cusp::blas::scal(cublas, x, 4.0f);

    ASSERT_EQUAL(x[0],  28.0);
    ASSERT_EQUAL(x[1],  20.0);
    ASSERT_EQUAL(x[2],  16.0);
    ASSERT_EQUAL(x[3], -12.0);
    ASSERT_EQUAL(x[4],   0.0);
    ASSERT_EQUAL(x[5],  16.0);

    View v(x);
    cusp::blas::scal(cublas, v, 2.0f);

    ASSERT_EQUAL(x[0],  56.0);
    ASSERT_EQUAL(x[1],  40.0);
    ASSERT_EQUAL(x[2],  32.0);
    ASSERT_EQUAL(x[3], -24.0);
    ASSERT_EQUAL(x[4],   0.0);
    ASSERT_EQUAL(x[5],  32.0);

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLASscal);

void TestCUBLASgemv(void)
{
    typedef cusp::device_memory MemorySpace;
    typedef typename cusp::array2d<float, MemorySpace>       Array2d;
    typedef typename cusp::array1d<float, MemorySpace>       Array1d;

    hipblasHandle_t handle;

    if(hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasCreate failed");
    }

    cusp::cublas::execution_policy cublas(handle);

    Array2d A;
    Array1d x(9);
    Array1d y(9);

    cusp::gallery::poisson5pt(A, 3, 3);

    x[0] =  7.0f;
    x[1] =  5.0f;
    x[2] =  4.0f;
    x[3] = -3.0f;
    x[4] =  0.0f;
    x[5] =  4.0f;
    x[6] = -3.0f;
    x[7] =  0.0f;
    x[8] =  4.0f;

    cusp::blas::gemv(cublas, A, x, y);

    ASSERT_EQUAL(y[0],  26.0);
    ASSERT_EQUAL(y[1],   9.0);
    ASSERT_EQUAL(y[2],   7.0);
    ASSERT_EQUAL(y[3], -16.0);
    ASSERT_EQUAL(y[4],  -6.0);
    ASSERT_EQUAL(y[5],   8.0);
    ASSERT_EQUAL(y[6],  -9.0);
    ASSERT_EQUAL(y[7],  -1.0);
    ASSERT_EQUAL(y[8],  12.0);

    if(hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
    {
      throw cusp::runtime_exception("hipblasDestroy failed");
    }
}
DECLARE_UNITTEST(TestCUBLASgemv);

